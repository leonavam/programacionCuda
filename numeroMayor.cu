#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define N 10

__global__ void numMayor(float *d_v, float *d_pos){


    float temp = 0,pos=0;
    for(int i=threadIdx.x; i<blockDim.x;i++){
        if(d_v[i] > temp){
            temp = d_v[i];
            pos = i;
        }
        
    }
    __syncthreads();
    if(pos>d_pos[threadIdx.x])
        d_pos[threadIdx.x] = pos;
    d_v[threadIdx.x] = temp;

}


int main(){

    float *h_pos, *h_v;
    float *d_pos, *d_v;

    int memSize = N*sizeof(float);
    srand(time(NULL));
    h_pos = (float*) malloc (memSize);
    h_v = (float*) malloc (memSize);
    hipMalloc((void**)&d_v, memSize);
    hipMalloc((void**)&d_pos, memSize);

    for(int i=0; i<N; i++)
        h_v[i] = (float) (rand()%11);
    
        
    for(int i=0; i<N;i++){
        printf("(%i)%.2f ",i, h_v[i]);
    }

    hipMemcpy(d_v, h_v, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_pos, h_pos, memSize, hipMemcpyHostToDevice);

    int block = 1;
    int thread = 10;
    numMayor<<<block,thread>>>(d_v,d_pos);
    
   
    hipMemcpy(h_pos,d_pos, memSize, hipMemcpyDeviceToHost);
    printf("\nPosicion %.2f -------\n", *h_pos);
    hipMemcpy(h_v,d_v, memSize, hipMemcpyDeviceToHost);
    printf(" Valor %.2f -------\n", *h_v);
    printf("---\n");
    for(int i=0; i<N;i++){
        printf("%.2f ", h_v[i]);
    }
    printf("\n---\n");
    for(int i=0; i<N;i++){
        printf("%.2f ", h_pos[i]);
    }
    return 0;
}