#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>

#define N 9

int main(){

    float *h_a, *h_b;
    float *d_a, *d_b;

    int memSize = N*sizeof(float);

    h_a = (float *) malloc (memSize);
    h_b = (float *) malloc (memSize);
    hipMalloc (&d_a, memSize);
    hipMalloc (&d_b, memSize);

    srand(time(NULL));
    for(int i=0; i<N; i++){
        h_a[i] = (float) (rand()%2);
    }

    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, d_a, memSize, hipMemcpyDeviceToDevice);
    hipMemcpy(h_b, d_b, memSize, hipMemcpyDeviceToHost);


    for(int i=0; i<N; i++){
        printf("%.2f - ",h_a[i]);
    }
    printf("\nEmpieza h_b\n");
    for(int i=0; i<N; i++){
        printf("%.2f - ",h_b[i]);
    }

    return 0;
}