#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void ecuaciones(int a, int b, int c, float *sol){

    int index = threadIdx.x; 
    float d = 0;
    float x=0, y=0;
    d = b*b-4*a*c;
    if (d > 0) {
        x = (-b+sqrt(d))/(2*a);
        y = (-b-sqrt(d))/(2*a);
        sol[index] = x;
        sol[index+1]=y;
    }
    else if (d == 0) {
        x = (-b)/(2*a);
        sol[index] = x;
    }
}


int main(){
    int a,b,c;
    float *h_x, *d_x;

    int memSize = 1*sizeof(float);
    h_x = (float *) malloc (memSize);
    hipMalloc(&d_x,memSize);

    printf("Ingrese a: ");
    scanf("%i", &a);
    printf("Ingrese b: ");
    scanf("%i", &b);
    printf("Ingrese c: ");
    scanf("%i", &c);

    ecuaciones<<<1,1>>> (a,b,c,d_x);
    hipMemcpy(h_x,d_x,sizeof(float),hipMemcpyDeviceToHost);

    printf("Respuesta x1: %f\n", h_x[0]);
    printf("Respuesta x2: %f\n", h_x[1]);

    char key = getchar();
    return 0;
}