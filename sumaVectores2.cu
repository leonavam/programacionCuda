#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#define N 9

__global__ void sumaVectores (float * d_a, float *d_b, float * d_c) {

        int index = blockIdx.x*blockDim.x+threadIdx.x;
        if (index < N )
                d_c[index] = d_a[index] +d_b[index];
}

int main () {

        float * h_a, *h_b, *h_c;
        float * d_a, *d_b, *d_c;

        int memsize = N*sizeof(float);

        h_a = (float *) malloc (memsize);
        h_b = (float *) malloc (memsize);
        h_c = (float *) malloc (memsize);
        hipMalloc (&d_a, memsize);
        hipMalloc (&d_b, memsize);
        hipMalloc (&d_c, memsize);

        for (int i=0; i<N; ++i) {
                h_a[i]=h_b[i] = 1.0f;
                h_c[i]= 0.0f;
        }

        hipMemcpy (d_a, h_a, memsize, hipMemcpyHostToDevice);
        hipMemcpy (d_b, h_b, memsize, hipMemcpyHostToDevice);
        hipMemcpy (d_c, h_c, memsize, hipMemcpyHostToDevice);

        int numBlocks =(int) ceil(N/2);
        int numThreads = 2;
        dim3 block (numBlocks);
        dim3 thread (numThreads);

        printf("Vector de %d elementos \nNumero de bloques = %d\nHilos por bloque = %d\n", N,numBlocks,numThreads);

        sumaVectores <<<block,thread>>> (d_a,d_b,d_c);

        hipMemcpy (h_c, d_c, memsize, hipMemcpyDeviceToHost);
        for (int i=0; i<N; ++i) {
                printf ("%f, ", h_c[i]);
        }

        printf ("\n");
        free (h_a);
        free (h_b);
        free (h_c);
        hipFree (d_a);
        hipFree (d_b);
        hipFree (d_c);

        return 0;



}
