#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 24

__global__ void muestraIndice(float *a, float *b, float *c){

    int global = blockIdx.x * blockDim.x + threadIdx.x;

    if(global < N){
        a[global] = threadIdx.x;
        b[global] = blockIdx.x;
        c[global] = global;
    }

}

int main(){

    float *h_hilo, *h_bloque, *h_global;
    float *d_hilo, *d_bloque, *d_global;

    int memSize = N * sizeof(float);

    h_hilo = (float*) malloc (memSize);
    h_bloque = (float*) malloc (memSize);
    h_global = (float*) malloc (memSize);
    hipMalloc(&d_bloque, memSize);
    hipMalloc(&d_hilo, memSize);
    hipMalloc(&d_global, memSize);

    int numBlock = 4;
    int numThread = 6;
    dim3 block(numBlock);
    dim3 thread(numThread);

    muestraIndice<<<block,thread>>>(d_hilo, d_bloque, d_global);

    hipMemcpy(h_hilo, d_hilo, memSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_bloque, d_bloque, memSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_global, d_global, memSize, hipMemcpyDeviceToHost);

    printf("Hilo\n");
    for(int i=0; i<N;i++){
        printf("%.2f ", h_hilo[i]);
    }
    printf("\nBloque\n");
    for(int i=0; i<N;i++){
        printf("%.2f ", h_bloque[i]);
    }
    printf("\nGlobal\n");
    for(int i=0; i<N;i++){
        printf("%.2f ", h_global[i]);
    }




    return 0;
}