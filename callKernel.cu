#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void sumaGPU(int a, int b, int *sol){

        *sol = a + b;
}

__host__ int sumaCPU(int a, int b){

    return (a+b);
}

int main(){

    int *h_sol;
    int *d_sol;
    int n1 = 3, n2 = 2, sol = 0;

    //Tamaño de reserva
    int memSize = (sizeof(int)); 

    //Reserva espacio en host
    h_sol = (int *) malloc (memSize);
    //Reserva espacio en GPU
    hipMalloc(&d_sol, memSize);

    //Llama funcion sumaCPU
    sol = sumaCPU(n1,n2);
    printf("Resultado CPU: %2i\n", sol);


    int numBlocks = 1;
    int numThreads = 1;
    dim3 block (numBlocks);
    dim3 thread (numThreads);

    sumaGPU <<<block,thread>>>(n1,n2,d_sol);

    //Recoge de GPU
    hipMemcpy(h_sol, d_sol, memSize, hipMemcpyDeviceToHost);
    printf("Resultado GPU: %2d\n", *h_sol);

    hipFree(d_sol);
    free(h_sol);

    //salida
    /*
    printf("Pulse INTRO para finalizar...");
    fflush(stdin);
    char key = getchar();*/
    return 0;
}