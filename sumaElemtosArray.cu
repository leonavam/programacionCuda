#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define TAM_V 1024
#define N_ThRead 128

__global__ void reduceVector(float *v1, float *v2, float *res){


    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int index2;

    for (int i = blockDim.x/2; i>=1; i=i/2){

        if(threadIdx.x < i){
            index2 = index + i;
            v1[index] += v1[index2];
        }
        __syncthreads();

    }
    if(threadIdx.x==0)
        res[blockIdx.x] = v1[index];

}

int main(){

    float *h_v1, *h_v2, *h_res;
    float *d_v1, *d_v2, *d_res;

    int memSize = TAM_V* sizeof(float);
//reserva en Host
    h_v1 = (float *) malloc (memSize);
    h_v2 = (float *) malloc (memSize);
    h_res = (float *) malloc (memSize);
//reserva en GPU
    hipMalloc(&d_v1, memSize);
    hipMalloc(&d_v2, memSize);
    hipMalloc(&d_res, memSize);

//inicializa vectores
    for (int i=0; i<TAM_V; i++){
        //h_v1[i] = h_v2[i] = (float) (rand()%2);
        h_v1[i] = h_v2[i] = 1.0f;
        h_res[i] = 0.0f;
    }

//copiar datos hacia GPU (device)
    hipMemcpy(d_v1, h_v1, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_v2, h_v2, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_res, h_res, memSize, hipMemcpyHostToDevice);

//Lanzar kernel
    int numBlocks = ceil(TAM_V/N_ThRead);
    int numThreads = N_ThRead;
    dim3 block (numBlocks);
    dim3 thread (numThreads);

    printf("Vector de %d elementos \nNumero de bloques = %d\nHilos por bloque = %d\n", TAM_V,numBlocks,numThreads);

//Calcula el tiempo de ejecucion
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

        reduceVector <<<block,thread>>>(d_v1,d_v2,d_res);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf ("El tiempo de ejecucion es %f msec\n", milliseconds);
    

    hipMemcpy(h_res, d_res, memSize, hipMemcpyDeviceToHost);

    float total =0.0f;
    for(int i=0; i<numBlocks; i++){
       total += h_res[i];
       printf("(%i)%.2f ",i,h_res[i]);
    }

    printf("\n-----%f\n",total);

    hipFree(d_v1);
    hipFree(d_v2);
    hipFree(d_res);
    free(h_v1);
    free(h_v2);
    free(h_res);


    return 0;
}